#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h> 
#include <assert.h>
#include <errno.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "cbir_cuda.h"
#include "cbir_cuda_kernel.cu"
//#include "cutil.h"
#include <sys/time.h>
#include <values.h>

#define THRESHOLD 0.00001

#define MIN_DOUBLE ((double) (1 << (sizeof(double) * 8 - 1)))//Minimum double
#define EPS 1.0e-10//Minimim distance between two numbers
#define ABS(a) (((a) < 0) ? -(a) : (a))

//static void Reshape(float *image, float* X, int elementos, int num_bands,
//		long int lines_samples) {
//	for (int i = 0; i < lines_samples; i++) {
//		for (int j = 0; j < num_bands; j++) {
//			X[j * elementos + i] = image[j * lines_samples + i];
//			//X[j*elementos+i]=1;
//		}
//	}
//}

static void LeerAleatorios(const char* random_filename, float *aleatorios,
		int n) {
	FILE *fp;
	float rdm;

	if ((fp = fopen(random_filename, "r")) == NULL) {
		printf(
				"ERROR %d. No se ha podido abrir el fichero de aleatorios: %s \n",
				errno, random_filename);
//		int status = system("PAUSE");
		exit(1);
	} else {
		fseek(fp, 0L, SEEK_CUR);
		int i = 0;
		while (i < n) {

			int result = fscanf(fp, "%f", &rdm);
			aleatorios[i] = rdm;
			i++;
		}

	}
	fclose(fp);
}

//static void checkCUDAError(const char *msg) {
//	hipError_t err = hipGetLastError();
//	if (hipSuccess != err) {
//		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
//		exit(-1);
//	}
//}

static void generarAleatorios(float *aleatorios, int n) {
	srand (time(NULL));
//	srand(10);
int	i;
//Generamos a con numeros aleatorios entre 0 y 1

	for (i = 0; i < n; i++) {
		//a[i]=drand48();
		aleatorios[i] = rand();
		//a[i]=1;
		//printf("%f\n", a[i]);
	}
}

//Function: GenerarAleatorios: Generates a set of n random numbers.
//Input: 	int *aleatorios: vector for store the random numbers.
//				int n: amount of random numbers.
//Output:	int *aleatorios:: vector of random numbers.
static void generarAleatoriosNfindr(int *aleatorios, int n,
		long int lines_samples) {
	srand (time(NULL));
//	srand(10);
for	(int i = 0; i < n; i++) {
		aleatorios[i] = rand() % lines_samples;
		//printf("Aleatorio %d = %d\n", i+1, aleatorios[i]); 
	}
}

//Function: LeerAleatorios: Reads random numbers set from a disk file.
//Input: 	char random_filename[200]:name of file with random numbers.
//				int n: amount of random numbers to read.
//				int *aleatorios: vector for store the random numbers.
//Output:	int *aleatorios:: vector of random numbers.
static void LeerAleatoriosNfindr(const char *random_filename, int *aleatorios,
		int n, int num_samples) {

	FILE *fp;
	int fila;
	int columna;
	int pixel = 0;

	if ((fp = fopen(random_filename, "r")) == NULL) {
		printf(
				"ERROR %d. No se ha podido abrir el fichero de aleatorios: %s \n",
				errno, random_filename);
//		int status = system("PAUSE");
		exit(1);
	} else {
		fseek(fp, 0L, SEEK_CUR);
		int i = 0;
		while (i < n) {
			int result = fscanf(fp, "%d", &fila);
			//   		printf("Fila= %d\n", fila);
			fila--;
			pixel += (fila * num_samples);

			result = fscanf(fp, "%d", &columna);
			//   		printf("Columna= %d\n", columna);
			columna--;
			pixel += columna;

			aleatorios[i] = pixel;
			i++;
			pixel = 0;
		}
	}
	fclose(fp);
}

//Function: Absoluto: computes the absolute value of a number.
//Input: 	double a: the number.
//Output:	double a: absolute value of the "a".
static double Absoluto(double a) {
	if (a < 0) {
		a = a * -1;
	}
	return a;
}
//Function: CambiarFilas: .
//Input: 	double *A:.
//				double *L:.
//				double *E:.
//				int n:.
//				int p:.
//Output:	:.
static void CambiarFilas(double* A, double* L, double* E, int n, int p) {

	int I;
	double max = -1;
	double elemento;
	for (int i = n; i < p; i++) {
		elemento = Absoluto(A[i * p + n]); //Columna n a partir de la fila n
		if (elemento > max) {
			max = elemento;
			I = i;
		}
	}

	double tmp;
	for (int i = 0; i < p; i++) {
		//Se cambia la fila I por la fila n
		tmp = A[n * p + i]; //elemento i de la fila n
		A[n * p + i] = A[I * p + i];
		A[I * p + i] = tmp;

		tmp = L[n * p + i]; //elemento i de la fila n
		L[n * p + i] = L[I * p + i];
		L[I * p + i] = tmp;

		E[n * p + i] = 0;
		E[I * p + i] = 0;
	}

	E[n * p + I] = 1;
	E[I * p + n] = 1;
}

//Function: LU: Makes the LU factorization of a  matrix .
//Input: 	double *L: L matrix of factorization.
//				double *U:U matrix of factorization.
//				double *Per: Permutations matrix.
//				int p:size of matrix.
//Output:	:L, U and Per.
// result == o: OK
// result != 0: error
static int LU(double *A, double *L, double *U, double *Per, int p) {

	double currentPivot;
	double maxPivot;
	double *E;
	double *PerAux;
	//double tmp;
	E = (double*) malloc(sizeof(double) * p * p);
	PerAux = (double*) malloc(sizeof(double) * p * p);

	for (int i = 0; i < p * p; i++) {
		Per[i] = 0;
		U[i] = 0;
		L[i] = 0;
	}
	for (int i = 0; i < p; i++) {
		Per[i * p + i] = 1;

	}
	for (int n = 0; n < p - 1; n++) {
		for (int i = 0; i < p * p; i++) {
			E[i] = 0;
		}
		for (int i = 0; i < p; i++) {
			E[i * p + i] = 1;
		}
		currentPivot = A[n * p + n];
		maxPivot = MIN_DOUBLE;
		double elemento;
		for (int i = n + 1; i < p; i++) {
			elemento = (A[i * p + n]); //Columna n a partir de la fila n
			if (elemento > maxPivot) {
				maxPivot = elemento;
			}
		}
		if (Absoluto(currentPivot) < EPS) { //zero, do row exchage always
			if (Absoluto(maxPivot) < EPS) { //not possible to exchange
				printf("unable to complete LU decomposition, bad A\n");
//				exit(-1);
				free(E);
				free(PerAux);
				return -1;
			} else {
				CambiarFilas(A, L, E, n, p);
			}
		} else {
			if (Absoluto(currentPivot) < Absoluto(maxPivot)) {
				CambiarFilas(A, L, E, n, p);
			}

		}

		for (int i = 0; i < p; i++) {
			for (int j = 0; j < p; j++) {
				PerAux[i * p + j] = 0;
				for (int k = 0; k < p; k++) {
					PerAux[i * p + j] += Per[i * p + k] * E[k * p + j];
				}
			}
		}
		memcpy(Per, PerAux, p * p * sizeof(double));
		for (int i = n + 1; i < p; i++) {
			L[i * p + n] = A[i * p + n] / A[n * p + n];
			A[i * p + n] = 0;
			for (int j = n + 1; j < p; j++) {
				A[i * p + j] = A[i * p + j] - L[i * p + n] * A[n * p + j];
			}
		}
	} //for n
	for (int i = 0; i < p; i++) {
		L[i * p + i]++;
	}
	for (int i = 0; i < p; i++) {
		for (int j = 0; j < p; j++) {
			Per[i * p + j] = PerAux[j * p + i];
		}
	}
	memcpy(U, A, p * p * sizeof(double));
	free(E);
	free(PerAux);
	return 0;
}

//Function: InvTri: Makes the inversion of a triangular matrix .
//Input: 	double *L: the triangular matrix.
//				int p:size of L matrix.
//Output:	double *L:inverse of L.
static void InvTri(double* L, int p) {
	double *I;
	I = (double*) calloc(p * p, sizeof(double));
	for (int i = 0; i < p; i++) {
		I[i * p + i] = 1;
	}

	for (int k = 0; k < p - 1; k++) {

		for (int i = k + 1; i < p; i++) {

			for (int j = 0; j <= k; j++) {
				L[i * p + j] -= L[k * p + i] * L[k * p + j];
				I[i * p + j] -= L[i * p + k] * I[k * p + j];
			}

		}

	}
	memcpy(L, I, p * p * sizeof(double));
	free(I);
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int getDeviceCount() {
	int deviceCount = -1;
	hipGetDeviceCount(&deviceCount);
	return deviceCount;
}

int initDevice(deviceIdentifier **deviceID, int device) {
	hipblasStatus_t status;
	*deviceID = (deviceIdentifier *) malloc(sizeof(deviceIdentifier));
	hipSetDevice(device);

	(*deviceID)->device = device;
	status = hipblasCreate(&((*deviceID)->handle));
	if (status != HIPBLAS_STATUS_SUCCESS) {
		free(*deviceID);
		return EXIT_FAILURE;
	}
	fprintf(stderr, "deviceIdentifier for device %d done!\n", (*deviceID)->device);
	fflush(stderr);
	return 0;
}

void destroyDevice(deviceIdentifier *deviceID) {
	hipblasDestroy((deviceID->handle));
	free(deviceID);
}

int spca(deviceIdentifier *deviceID, const float* h_image, int num_lines,
		int num_samples, int num_bands, int lines_samples, int n_pc,
		int generate, const char* random_vector_file, int fixed_n_iterations,
		int n_iterations, float* h_B) {
	hipblasStatus_t status;
	hipError_t error;

	hipSetDevice(deviceID->device);
	int iterations;
	float max;
	int max_i;
	float aux;

	// Pointers to host memory
	float *h_EIGEN;
	float *h_EIGENold;
	float *h_P;
	float *h_Pcoeffs;
	float *h_defl;
	float *h_sumFi;
	float *h_Fitmp;
	float *h_Ytmp_aux;
	float *h_P_aux;

	float *d_X;
	float *d_P;
	float *d_Y;
	float *d_Y2;
	float *d_Pcoeffs;
	float *d_defl;
	float *d_sumFi;
	float *d_Ytmp;
	float *d_Fitmp;
	float *d_Ytmp_aux;
	float *d_pixel;
	float *d_P_aux;
	float *d_B;
	float *d_XX;
	float *d_deltaP;

	h_EIGEN = (float*) calloc(n_pc, sizeof(float));
	h_EIGENold = (float*) calloc(n_pc, sizeof(float));
	h_P = (float*) malloc(num_bands * n_pc * sizeof(float));
	h_Pcoeffs = (float*) malloc(n_pc * n_pc * sizeof(float));
	h_defl = (float*) malloc(n_pc * n_pc * sizeof(float));
	h_sumFi = (float*) malloc(n_pc * num_bands * sizeof(float));
	h_Fitmp = (float*) malloc(n_pc * num_bands * sizeof(float));
	h_Ytmp_aux = (float*) malloc(n_pc * n_pc * sizeof(float));
	h_P_aux = (float*) malloc(n_pc * n_pc * sizeof(float));

	/* Allocate device memory for the matrices */
	error = hipMalloc((void**) &d_X,
			lines_samples * num_bands * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (X)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_P, num_bands * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (P)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Y, lines_samples * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Y)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Y2, lines_samples * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Y2)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Pcoeffs, n_pc * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Pcoeffs)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_defl, n_pc * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (defl)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_sumFi, n_pc * num_bands * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (sumFi)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Ytmp, n_pc * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Ytmp)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Fitmp, n_pc * num_bands * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Fitmp)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_Ytmp_aux, n_pc * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (Ytmp_aux)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_pixel, num_bands * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (pixel)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_P_aux, n_pc * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (P_aux)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_B, lines_samples * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (B)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_XX, num_bands * num_bands * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (XX)\n");
		return EXIT_FAILURE;
	}

	error = hipMalloc((void**) &d_deltaP, num_bands * n_pc * sizeof(float));
	if (error != hipSuccess) {
		fprintf(stderr, "!!!! device memory allocation error (deltaP)\n");
		return EXIT_FAILURE;
	}

	if (generate == 0) {
		//Random vector reading
		LeerAleatorios(random_vector_file, h_P, num_bands * n_pc);
	} else if (generate == 1) {
		generarAleatorios(h_P, num_bands * n_pc);
		//printf("Se generan los números aleatorios\n");
	}

	hipDeviceSynchronize();

	status = hipblasSetVector(num_lines * num_samples * num_bands, sizeof(float),
			h_image, 1, d_X, 1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device access error (write X)\n");
		return EXIT_FAILURE;
	}

	int num_blocks_NormalizeX = num_bands;
	//printf("num_blocks_NormalizeX %d\n", num_blocks_NormalizeX);

	int num_threads_NormalizeX = 512;
	//printf("num_threads_NormalizeX %d\n", num_threads_NormalizeX);

	iterations = (int) ceil(
			((float) lines_samples / (float) num_threads_NormalizeX));
	//printf("Iterations NormalizeX %d\n", iterations);

	NormalizeX<<<num_blocks_NormalizeX, num_threads_NormalizeX>>>(d_X, d_pixel, num_bands, num_lines, num_samples, iterations);

	hipDeviceSynchronize();

	status = hipblasSetVector(num_bands * n_pc, sizeof(float), h_P, 1, d_P, 1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device access error (write P)\n");
		return EXIT_FAILURE;
	}

	/* P_aux= P' * P */
	float alpha = 1;
	float beta = 0;
	status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_pc, n_pc,
			num_bands, &alpha, d_P, num_bands, d_P, num_bands, &beta, d_P_aux,
			n_pc);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! kernel execution error (P_aux= P' * P).\n");
		return EXIT_FAILURE;
	}

	/* diag(diag(P_aux))^-0.5 */
	status = hipblasGetVector(n_pc * n_pc, sizeof(float), d_P_aux, 1, h_P_aux,
			1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device access error (read P_aux)\n");
		return EXIT_FAILURE;
	}

	for (int i = 0; i < n_pc; i++) { //columna
		for (int j = 0; j < n_pc; j++) { //fila
			if (j != i) {
				h_P_aux[(i * n_pc) + j] = 0;
			} else {
				h_P_aux[(i * n_pc) + j] = pow(h_P_aux[(i * n_pc) + j], -0.5);
			}
		}
	}

	status = hipblasSetVector(n_pc * n_pc, sizeof(float), h_P_aux, 1, d_P_aux,
			1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device access error (write P_aux)\n");
		return EXIT_FAILURE;
	}

	/* P = P * (diag(diag(P'*P))^-0.5) */
	alpha = 1;
	beta = 0;
	status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, num_bands,
			n_pc, n_pc, &alpha, d_P, num_bands, d_P_aux, n_pc, &beta, d_P,
			num_bands);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr,
				"!!!! kernel execution error (P = P * (diag(diag(P'*P))^-0.5)).\n");
		return EXIT_FAILURE;
	}

	/* XX = X'*X */
	alpha = 1;
	beta = 0;
	status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, num_bands,
			num_bands, lines_samples, &alpha, d_X, lines_samples, d_X,
			lines_samples, &beta, d_XX, num_bands);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! kernel execution error (XX = X'*X.\n");
		return EXIT_FAILURE;
	}

	hipDeviceSynchronize();

	int iter;
	if (fixed_n_iterations == 1) {
		//printf("Numero de iteraciones fijadas a %d\n", n_iterations);
	}

	for (iter = 0; iter < n_iterations; iter++) {

		/*Compute deflation matrix*/
		/* Pcoeffs = P' * P */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_pc,
				n_pc, num_bands, &alpha, d_P, num_bands, d_P, num_bands, &beta,
				d_Pcoeffs, n_pc);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (Pcoeffs = P' * P ).\n");
			return EXIT_FAILURE;
		}
		//
		hipDeviceSynchronize();

		/* defl = -Pcoeffs*/
		hipMemcpy(h_Pcoeffs, d_Pcoeffs, n_pc * n_pc * sizeof(float),
				hipMemcpyDeviceToHost);

		for (int i = 0; i < n_pc * n_pc; i++) {
			h_defl[i] = -h_Pcoeffs[i];
		}

		/* defl = triu(defl,+1) + eye(size(defl)) */
		for (int i = 0; i < n_pc; i++) { //col
			for (int j = 0; j < n_pc; j++) { //fil
				if (i == j) {
					h_defl[(i * n_pc) + j] = 1;
				} else {
					if (j > i) {
						h_defl[(i * n_pc) + j] = 0;
					}
				}
			}
		}

		for (int i = 0; i < n_pc; i++) {
			for (int j = i + 2; j < n_pc; j++) {
				for (int k = i + 2; k <= j; k++) {
					h_defl[i + (j * n_pc)] -= h_defl[i + ((k - 1) * n_pc)]
							* h_Pcoeffs[(k - 1) + (j * n_pc)];
				}
			}
		}
		status = hipblasSetVector(n_pc * n_pc, sizeof(float), h_defl, 1, d_defl,
				1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (write defl)\n");
			return EXIT_FAILURE;
		}

		hipDeviceSynchronize();

		/* deltaP = P * defl */

		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				num_bands, n_pc, n_pc, &alpha, d_P, num_bands, d_defl, n_pc,
				&beta, d_deltaP, num_bands);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (deltaP = P * defl ).\n");
			return EXIT_FAILURE;
		}

		hipDeviceSynchronize();

		/* sumFi = deltaP' * XX */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_pc,
				num_bands, num_bands, &alpha, d_deltaP, num_bands, d_XX,
				num_bands, &beta, d_sumFi, n_pc);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (sumFi = deltaP' * XX).\n");
			return EXIT_FAILURE;
		}

		hipDeviceSynchronize();

		/* Ytmp = sumFi * deltaP */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n_pc,
				n_pc, num_bands, &alpha, d_sumFi, n_pc, d_deltaP, num_bands,
				&beta, d_Ytmp, n_pc);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (Ytmp = sumFi * deltaP).\n");
			return EXIT_FAILURE;
		}

		hipDeviceSynchronize();

		hipMemcpy(h_Ytmp_aux, d_Ytmp, n_pc * n_pc * sizeof(float),
				hipMemcpyDeviceToHost);

		for (int i = 0; i < n_pc; i++) { //col
			for (int j = 0; j < n_pc; j++) { //fil
				if (i == j) {
					//Aprovechamos para formar EIGEN a partir de la diagonal de Ytmp
					h_EIGEN[i] = h_Ytmp_aux[(i * n_pc) + j];
					//printf("EIGEN[%d] = %f\n", i, h_EIGEN[i]);
				}
				if (i >= j) {
					h_Ytmp_aux[(i * n_pc) + j] = 0;
				}

			}
		}
		status = hipblasSetVector(n_pc * n_pc, sizeof(float), h_Ytmp_aux, 1,
				d_Ytmp_aux, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (write Ytmp_aux)\n");
			return EXIT_FAILURE;
		}

		/* Fitmp = Ytmp_aux * P' */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n_pc,
				num_bands, n_pc, &alpha, d_Ytmp_aux, n_pc, d_P, num_bands,
				&beta, d_Fitmp, n_pc);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (Fitmp = Ytmp_aux * P').\n");
			return EXIT_FAILURE;
		}
		//
		hipDeviceSynchronize();

		/* sumFi = sumFi - Fitmp */
		status = hipblasGetVector(n_pc * num_bands, sizeof(float), d_sumFi, 1,
				h_sumFi, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (read sumFi)\n");
			return EXIT_FAILURE;
		}
		status = hipblasGetVector(n_pc * num_bands, sizeof(float), d_Fitmp, 1,
				h_Fitmp, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (read Fitmp)\n");
			return EXIT_FAILURE;
		}

		for (int i = 0; i < n_pc * num_bands; i++) {
			h_sumFi[i] -= h_Fitmp[i];
		}
		hipDeviceSynchronize();

		/* P = sumFi ' */
		for (int i = 0; i < num_bands; i++) {
			for (int j = 0; j < n_pc; j++) {
				h_P[(j * num_bands) + i] = h_sumFi[(i * n_pc) + j];
			}
		}

		status = hipblasSetVector(n_pc * num_bands, sizeof(float), h_sumFi, 1,
				d_sumFi, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (write sumFi)\n");
			return EXIT_FAILURE;
		}

		status = hipblasSetVector(num_bands * n_pc, sizeof(float), h_P, 1, d_P,
				1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (write P)\n");
			return EXIT_FAILURE;
		}

		hipDeviceSynchronize();

		/* P_aux= P' * P */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n_pc,
				n_pc, num_bands, &alpha, d_P, num_bands, d_P, num_bands, &beta,
				d_P_aux, n_pc);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! kernel execution error (P_aux= P' * P).\n");
			return EXIT_FAILURE;
		}

		/* diag(diag(P_aux))^-0.5 */
		status = hipblasGetVector(n_pc * n_pc, sizeof(float), d_P_aux, 1,
				h_P_aux, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (read P_aux)\n");
			return EXIT_FAILURE;
		}

		for (int i = 0; i < n_pc; i++) { //columna
			for (int j = 0; j < n_pc; j++) { //fila
				if (j != i) {
					h_P_aux[(i * n_pc) + j] = 0;
				} else {
					h_P_aux[(i * n_pc) + j] = pow(h_P_aux[(i * n_pc) + j],
							-0.5);
				}
			}
		}

		status = hipblasSetVector(n_pc * n_pc, sizeof(float), h_P_aux, 1,
				d_P_aux, 1);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr, "!!!! device access error (write P_aux)\n");
			return EXIT_FAILURE;
		}

		/* P = P * (diag(diag(P'*P))^-0.5) */
		alpha = 1;
		beta = 0;
		status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
				num_bands, n_pc, n_pc, &alpha, d_P, num_bands, d_P_aux, n_pc,
				&beta, d_P, num_bands);
		if (status != HIPBLAS_STATUS_SUCCESS) {
			fprintf(stderr,
					"!!!! kernel execution error (P = P * (diag(diag(P'*P))^-0.5)).\n");
			return EXIT_FAILURE;
		}
		//
		hipDeviceSynchronize();

		/* EIGEN = diag(Ytmp) */
		//ya está hecho
		max = -1;
		max_i = -1;
		for (int i = 0; i < n_pc; i++) {
			//aux=ABS(h_EIGEN[i]-h_EIGENold[i])/ABS(h_EIGENold[i]);
			//aux=ABS(h_EIGEN[i]-h_EIGENold[i]);
			aux = ABS(h_EIGEN[i]-h_EIGENold[i]) / ABS(h_EIGENold[i]);
			//printf("aux = %f\n", aux);
			if (max < aux) {
				max = aux;
				max_i = i;
			}
		}
		//printf("max = %f\n", max);
		//printf("max_i = %d\n", max_i);

		if (fixed_n_iterations == 0) {
			if (max < THRESHOLD
					|| ABS(h_EIGEN[max_i]-h_EIGENold[max_i]) < THRESHOLD) {
				//printf("ITER = %d\n", iter);
				break;
			}
		}
		for (int i = 0; i < n_pc; i++) {
			h_EIGENold[i] = h_EIGEN[i];
			//if(iter==87 || iter==88 || iter==89){
			//printf("EIGENold[%d] = %f\n", i, h_EIGENold[i]);
			//}
		}
	}

	//printf("ITER\t%d\n", iter);

	/* B = X * P */
	alpha = 1;
	beta = 0;
	status = hipblasSgemm(deviceID->handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
			lines_samples, n_pc, num_bands, &alpha, d_X, lines_samples, d_P,
			num_bands, &beta, d_B, lines_samples);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! kernel execution error (B = X * P)\n");
		return EXIT_FAILURE;
	}
	hipDeviceSynchronize();

	status = hipblasGetVector(lines_samples * n_pc, sizeof(float), d_B, 1, h_B,
			1);
	if (status != HIPBLAS_STATUS_SUCCESS) {
		fprintf(stderr, "!!!! device access error (read B)\n");
		return EXIT_FAILURE;
	}

	free(h_EIGEN);
	free(h_EIGENold);
	free(h_P);
	free(h_Pcoeffs);
	free(h_defl);
	free(h_sumFi);
	free(h_Fitmp);
	free(h_Ytmp_aux);
	free(h_P_aux);

	hipFree(d_X);
	hipFree(d_P);
	hipFree(d_Y);
	hipFree(d_Y2);
	hipFree(d_Pcoeffs);
	hipFree(d_defl);
	hipFree(d_sumFi);
	hipFree(d_Ytmp);
	hipFree(d_Fitmp);
	hipFree(d_Ytmp_aux);
	hipFree(d_pixel);
	hipFree(d_P_aux);
	hipFree(d_B);
	hipFree(d_XX);
	hipFree(d_deltaP);

	hipDeviceSynchronize();
	return 0;
}

int nfindr(deviceIdentifier *deviceID, const float* h_image,
		const int num_samples, const int n_pc, const int lines_samples,
		const int g_aleatorios, const char* nfinder_init_file, int *P) {
	hipSetDevice(deviceID->device);

	int p = n_pc + 1;		//Numbers of reduced image principal components + 1.

	int *aleatorios;

	double *MatrixTest;
	double *MatrixTestLU;
	double *matrix;
	double volumeactual = 0;
	int maxit;
	int it;
	double v1, v2;

	int bloques_reduccion;
	//int hilos_reduccion=atoi(argv[13]);
	int hilos_reduccion = 512;
	bloques_reduccion = ceil(
			(double) lines_samples / (double) 2 / (double) hilos_reduccion);

	//printf ("bloques_reduccion = %d\n", bloques_reduccion);
	//printf ("hilos_reduccion = %d\n", hilos_reduccion);

	//int hilos_multiplicacion=atoi(argv[14]);
	int hilos_multiplicacion = 512;
	int bloques_multiplicacion;
	bloques_multiplicacion = ceil(
			(double) lines_samples / (double) hilos_multiplicacion);
	//printf ("hilos_multiplicacion = %d\n", hilos_multiplicacion);
	//printf ("bloques_multiplicacion = %d\n", bloques_multiplicacion);
	int elementos_Vvolume = hilos_reduccion * bloques_reduccion * 2;
	//num_ceros=elementos_Vvolume-lines_samples;

	double *HIM2x2c;
	double *HIM2x2;

	double *pixelactual;

	double *Ldet;
	double *Udet;
	double *Pdet;
	double sUdet;

	double *aux2;
	bool comp = false;
	//double tmp;
	double tmp2;

	aleatorios = (int*) malloc(sizeof(int) * p);
	MatrixTest = (double*) malloc(sizeof(double) * p * p);
	MatrixTestLU = (double*) malloc(sizeof(double) * p * p);
	matrix = (double*) malloc(sizeof(double) * p * p);

	HIM2x2c = (double*) calloc(lines_samples * p, sizeof(double));
	HIM2x2 = (double*) malloc(lines_samples * p * sizeof(double));

	pixelactual = (double*) malloc(sizeof(double) * n_pc);

	Ldet = (double*) malloc(p * p * sizeof(double));
	Udet = (double*) malloc(p * p * sizeof(double));
	Pdet = (double*) malloc(p * p * sizeof(double));

	aux2 = (double*) malloc(p * sizeof(double));

	///////////////////////////////////////////////////////////////////////////////////////////

	double *d_aux2;
	double *d_HIM2x2;
	double *d_Vvolume;

	int *h_indices;
	int *d_indices;

	double *h_volumenes;
	double *d_volumenes;

	hipMalloc((void**) &d_aux2, (p * sizeof(double)));
	hipMalloc((void**) &d_HIM2x2, (lines_samples * p * sizeof(double)));
	hipMalloc((void**) &d_Vvolume, (elementos_Vvolume * sizeof(double)));

	h_indices = (int*) malloc(bloques_reduccion * sizeof(int));
	hipMalloc((void**) &d_indices, (bloques_reduccion * sizeof(int)));
	hipMemcpy(d_indices, h_indices, (bloques_reduccion * sizeof(int)),
			hipMemcpyHostToDevice);

	h_volumenes = (double*) malloc(bloques_reduccion * sizeof(double));
	hipMalloc((void**) &d_volumenes, (bloques_reduccion * sizeof(double)));
	hipMemcpy(d_volumenes, h_volumenes, (bloques_reduccion * sizeof(double)),
			hipMemcpyHostToDevice);

	if (g_aleatorios == 0) {
		LeerAleatoriosNfindr(nfinder_init_file, aleatorios, p, num_samples);
	} else {
		generarAleatoriosNfindr(aleatorios, p, lines_samples);
	}

	//Make the initial matrix
	for (int k = 0; k < p; k++) {
		P[k] = aleatorios[k];
		MatrixTest[k] = 1;
		matrix[k] = 1;
		for (int i = 1; i < p; i++) {	//from 1 to 19
			MatrixTest[k + (i * p)] = (double) h_image[P[k]
					+ ((i - 1) * lines_samples)];
			matrix[k + (i * p)] = (double) h_image[P[k]
					+ ((i - 1) * lines_samples)];
		}
	}

	hipDeviceSynchronize();

	//Compute initial matrix volume
	for (int k = 0; k < p - 1; k++) {
		for (int i = k + 1; i < p; i++) {
			for (int j = k + 1; j < p; j++) {
				matrix[p * i + j] -= matrix[p * i + k] * matrix[p * k + j]
						/ matrix[p * k + k];
			}
		}
	}
	double deter = 1.0;
	for (int i = 0; i < p; i++) {
		deter *= matrix[p * i + i];
	}

	volumeactual = Absoluto(deter);

	hipDeviceSynchronize();
	//printf("VOLUMEN INICIAL %e\n", volumeactual);

	maxit = 3 * p;
	it = 1;
	v1 = -1;
	v2 = volumeactual;

	for (int i = 0; i < lines_samples; i++) {
		HIM2x2c[i] = 1;
	}
	for (int i = 0; i < lines_samples * n_pc; i++) {
		HIM2x2c[lines_samples + i] = (double) h_image[i];
	}

	while ((it <= maxit) && (v2 > v1)) {
		memcpy(HIM2x2, HIM2x2c, lines_samples * p * sizeof(double));
		for (int k = 0; k < p; k++) {
			for (int i = 0; i < n_pc; i++) {//Change the endmember k for the endmember p
				pixelactual[i] = MatrixTest[(i + 1) * p + k];
				MatrixTest[(i + 1) * p + k] = MatrixTest[(i + 1) * p + (p - 1)];
				MatrixTest[(i + 1) * p + (p - 1)] = pixelactual[i];
			}
			for (int i = 0; i < p; i++) {	//Set the last column 0 0 0 ... 1
				MatrixTest[i * p + (p - 1)] = 0;
			}
			MatrixTest[p * p - 1] = 1;
			memcpy(MatrixTestLU, MatrixTest, p * p * sizeof(double));//Not to overwrite MatrixTest doing LU
			if (LU(MatrixTestLU, Ldet, Udet, Pdet, p) != 0) {
				free(h_indices);
				free(h_volumenes);

				hipFree(d_aux2);
				hipFree(d_HIM2x2);
				hipFree(d_Vvolume);
				hipFree(d_indices);
				hipFree(d_volumenes);

				hipDeviceSynchronize();
				return -1;
			}
			sUdet = 1;
			for (int i = 0; i < p; i++) {
				sUdet *= Udet[i * p + i];
			}
			sUdet = Absoluto(sUdet);

			InvTri(Ldet, p);	//inv(Ldet)

			for (int j = 0; j < p; j++) {
				aux2[j] = 0;
				for (int k = 0; k < p; k++) {
					aux2[j] += Ldet[(p - 1) * p + k] * Pdet[k * p + j];
				}
			}

			hipMemcpy(d_aux2, aux2, (p * sizeof(double)),
					hipMemcpyHostToDevice);
			if (k == 0) {
				hipMemcpy(d_HIM2x2, HIM2x2,
						(p * lines_samples * sizeof(double)),
						hipMemcpyHostToDevice);
			}
			tmp2 = (sUdet / Udet[p * p - 1]);

			VolumeCalculation<<<bloques_multiplicacion, hilos_multiplicacion>>>(d_aux2, d_HIM2x2, d_Vvolume, tmp2, lines_samples, p);
			Reduction_vol<<<bloques_reduccion, hilos_reduccion>>>(d_Vvolume, d_volumenes, d_indices, lines_samples);

			hipMemcpy(h_volumenes, d_volumenes,
					(bloques_reduccion * sizeof(double)),
					hipMemcpyDeviceToHost);
			hipMemcpy(h_indices, d_indices, (bloques_reduccion * sizeof(int)),
					hipMemcpyDeviceToHost);

			for (int i = 0; i < bloques_reduccion; i++) {
				//printf("--------- = %e\n", h_volumenes[i]);
				if (h_volumenes[i] > volumeactual) {
					volumeactual = h_volumenes[i];
					P[k] = h_indices[i];
					comp = true;
				}
			}
			if (comp == true) {
				//printf("---> update with pixel @ %d | abs(det(E))= %e\n", P[k], Vvolume[P[k]]);
				for (int j = 0; j < n_pc; j++) {
					pixelactual[j] = h_image[j * lines_samples + P[k]];
				}
				comp = false;
			}
			for (int i = 0; i < n_pc; i++) {
				MatrixTest[(i + 1) * p + (p - 1)] = MatrixTest[(i + 1) * p + k];
				MatrixTest[(i + 1) * p + k] = pixelactual[i];
			}
		}	//for k

		//for(int i=0; i<p; i++){
		//	printf("Pixel %d -> [%d, %d]\n", i, P[i]/num_samples, P[i]%num_samples);
		//}

		it++;
		v1 = v2;
		v2 = volumeactual;
	}		//while and

	int nit = it - 1;
	if (nit < maxit) {
		//printf("End, convergence @ iteration [%d]. Final abs(det(E)) = %e\n", nit, volumeactual);
		//printf("NFINDR_IT\t%d\n", nit);
	} else {
		printf(
				"End, NO convergence until iteration # %d. the abs(det(E)) = %e\n",
				nit, volumeactual);
	}
	//printf("The NFINDR found solution is:\n");
	for (int i = 0; i < p; i++) {
		//printf("Pixel %d -> [%d, %d]\n", i, P[i]/num_samples, P[i]%num_samples);
	}

	/*		FILE *f_end = fopen(argv[10],"w+");*/
	/*		for(int i=0; i<p; i++){*/
	/*			fprintf(f_end, "===== Endmembers %d =====\n", i);*/
	/*			for(int j=0; j<num_bands_orig; j++){*/
	/*				fprintf(f_end, "%f\n", h_X[P[i]+(lines_samples*j)]);*/
	/*			}*/
	/*		}*/
	/*		fclose(f_end);*/

	free(h_indices);
	free(h_volumenes);

	hipFree(d_aux2);
	hipFree(d_HIM2x2);
	hipFree(d_Vvolume);
	hipFree(d_indices);
	hipFree(d_volumenes);

	hipDeviceSynchronize();
	return 0;
}

void lsu(deviceIdentifier *deviceID, const float* h_image, const int* P,
		const int num_lines, const int num_samples, const int num_bands,
		const int lines_samples, const int num_endmembers,
		float* abundance_map) {

	hipSetDevice(deviceID->device);

	//Pointers to Host memory
	double *h_end;
	double *h_endt;
	double *h_etxe;
	double *h_etxei;
	double *h_matriz_computo;
	float *h_matriz_computo2;

	//Pointers to Device memory
	float *d_imagen;
	float *d_matriz_computo;
	float *d_imagen_unmixing;

	//Memory assignmrnt for Device and Host
	h_end = (double*) malloc(num_bands * num_endmembers * sizeof(double));//Matriz de Endmembers
	h_endt = (double*) malloc(num_bands * num_endmembers * sizeof(double));	//Matriz de Endmembers Traspuesta
	h_etxe = (double*) malloc(num_endmembers * num_endmembers * sizeof(double));// h_endt * h_end
	h_etxei = (double*) malloc(
			num_endmembers * num_endmembers * sizeof(double));// Inversa(h_etxe)
	h_matriz_computo = (double*) malloc(
			num_endmembers * num_bands * sizeof(double));	// h_etxei * h_endt
	h_matriz_computo2 = (float*) malloc(
			num_endmembers * num_bands * sizeof(float));

	//Copy CPU -> GPU
	hipMalloc((void**) &d_imagen,
			(num_lines * num_samples * num_bands * sizeof(float)));
	hipMalloc((void**) &d_matriz_computo,
			(num_endmembers * num_bands * sizeof(float)));
	hipMalloc((void**) &d_imagen_unmixing,
			(num_lines * num_samples * num_endmembers * sizeof(float)));

	int fila, columna;
	//FILE *fpe;
	//fpe = fopen(argv[9], "w");
	for (int k = 0; k < num_endmembers; k++) {
		//fprintf(fpe,"======== Endmember %d ========\n", k);
		for (int l = 0; l < num_bands; l++) {
			//fprintf(fpe,"%f\n", h_imagen[l*lines_samples+P[k]]);
			h_endt[k * num_bands + l] = h_image[l * lines_samples + P[k]];

			fila = (k * num_bands + l) / num_bands;

			columna = (k * num_bands + l) % num_bands;

			h_end[columna * num_endmembers + fila] = h_endt[fila * num_bands
					+ columna];
		}
	}
	//fclose(fpe);
	hipDeviceSynchronize();

	//h_endt x h_end = h_etxe
	for (int i = 0; i < num_endmembers; i++) {
		for (int j = 0; j < num_endmembers; j++) {
			h_etxe[i * num_endmembers + j] = 0;
			for (int k = 0; k < num_bands; k++) {
				h_etxe[i * num_endmembers + j] += h_endt[i * num_bands + k]
						* h_end[k * num_endmembers + j];
				//printf("%f\n", h_etxe[i*N_END+j]);
				//system("PAUSE");
			}
		}
	}

	hipDeviceSynchronize();

	//h_etxei = inv(h_etxe)
	double *b;
	//float *c;
	int n = num_endmembers;
	b = (double*) malloc(num_endmembers * num_endmembers * sizeof(double));	//matriz de los términos independientes
	//c = (float*) malloc (N_END * N_END * sizeof(float));
	for (int i = 0; i < num_endmembers * num_endmembers; i++) {
		b[i] = 0;
		h_etxei[i] = 0;
		//c[i]=h_etxe[i];
	}
	//identity matrix
	for (int i = 0; i < n; i++) {
		b[i * num_endmembers + i] = 1.0;
	}

	//Matrix and independent terms transformation
	for (int k = 0; k < n - 1; k++) {
		for (int i = k + 1; i < n; i++) {
			//independent terms
			for (int s = 0; s < n; s++) {
				b[i * num_endmembers + s] -= h_etxe[i * num_endmembers + k]
						* b[k * num_endmembers + s]
						/ h_etxe[k * num_endmembers + k];
			}

			//matrix elements
			for (int j = k + 1; j < n; j++) {
				h_etxe[i * num_endmembers + j] -= h_etxe[i * num_endmembers + k]
						* h_etxe[k * num_endmembers + j]
						/ h_etxe[k * num_endmembers + k];
			}
		}
	}

	//unknown calculation, inverse matrix elements
	for (int s = 0; s < n; s++) {
		h_etxei[(n - 1) * num_endmembers + s] = b[(n - 1) * num_endmembers + s]
				/ h_etxe[(n - 1) * num_endmembers + n - 1];
		for (int i = n - 2; i >= 0; i--) {
			h_etxei[i * num_endmembers + s] = b[i * num_endmembers + s]
					/ h_etxe[i * num_endmembers + i];
			for (int k = n - 1; k > i; k--) {
				h_etxei[i * num_endmembers + s] -=
						h_etxe[i * num_endmembers + k]
								* h_etxei[k * num_endmembers + s]
								/ h_etxe[i * num_endmembers + i];
			}
		}
	}

	hipDeviceSynchronize();

	//h_etxei x h_endt = h_matriz_computo
	for (int i = 0; i < num_endmembers; i++) {
		for (int j = 0; j < num_bands; j++) {
			h_matriz_computo[i * num_bands + j] = 0;
			for (int k = 0; k < num_endmembers; k++) {
				h_matriz_computo[i * num_bands + j] += h_etxei[i
						* num_endmembers + k] * h_endt[k * num_bands + j];
			}
		}
	}

	for (int i = 0; i < num_endmembers; i++) {
		for (int j = 0; j < num_bands; j++) {
			h_matriz_computo2[i * num_bands + j] = (float) h_matriz_computo[i
					* num_bands + j];
		}
	}

	hipDeviceSynchronize();

	//Copy CPU -> GPU
	hipMemcpy(d_imagen, h_image,
			(num_lines * num_samples * num_bands * sizeof(float)),
			hipMemcpyHostToDevice);
	// Timo: no need to copy this one: it will only contain result values.
	//	hipMemcpy(d_imagen_unmixing, abundance_map,
	//			(num_lines * num_samples * num_endmembers * sizeof(float)),
	//			hipMemcpyHostToDevice);
	hipMemcpy(d_matriz_computo, h_matriz_computo2,
			(num_endmembers * num_bands * sizeof(float)),
			hipMemcpyHostToDevice);

	//int num_bloques_lsu = atoi(argv[11]);
	//int num_hilos_lsu = atoi(argv[12]);

	int num_hilos_lsu = 512;
	int num_bloques_lsu = (int) ceil(
			((float) lines_samples / (float) num_hilos_lsu));
	//printf ("num_bloques_lsu = %d\n", num_bloques_lsu);
	//printf ("num_hilos_lsu = %d\n", num_hilos_lsu);

	//Kernel execution
	Unmixing<<<num_bloques_lsu, num_hilos_lsu>>>(d_imagen, d_imagen_unmixing, d_matriz_computo, num_lines, num_samples, num_bands, num_endmembers);

	hipDeviceSynchronize();

	//Copy GPU -> CPU
	hipMemcpy(abundance_map, d_imagen_unmixing,
			(num_lines * num_samples * num_endmembers * sizeof(float)),
			hipMemcpyDeviceToHost);

	hipFree(d_imagen);
	hipFree(d_matriz_computo);
	hipFree(d_imagen_unmixing);

	hipDeviceSynchronize();
}
