#include "hip/hip_runtime.h"

#define N  30
#define TAMANIO_MATRIZ_C 224
#define BLOCKSIZE_MEDIA 512

#define ABS(a)	   (((a) < 0) ? -(a) : (a))

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
//											Kernels SPCA							  //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////

__global__ void NormalizeX(float* d_image, float *d_pixel, int num_bands, int num_lines, int num_samples, int iterations){
	__shared__ float sdata[BLOCKSIZE_MEDIA];
	__shared__ float smean[1];
	__shared__ float svar[1];
	int it, s;
	unsigned int tid = threadIdx.x;
	int element;
	if(tid==0){
		smean[0]=0;
		svar[0]=0;
	}
	
	for (it=0; it<iterations; it++){
		element=(num_lines*num_samples*blockIdx.x)+(blockDim.x*it);
		if((it*blockDim.x)+tid<num_lines*num_samples){
			sdata[tid]=d_image[element+tid];
		}
		else{
			sdata[tid]=0;
		}
		__syncthreads();
		
		for(s=blockDim.x/2; s>0; s=s/2){
			if (tid < s){
				sdata[tid]+=sdata[tid+s];
			}
			__syncthreads();
		}
		
		if(tid==0){
			smean[0]+=sdata[0];
		}
		__syncthreads();
				
	}
	if(tid==0){
		smean[0]/=(num_lines*num_samples);
		
	}
	__syncthreads();
	
	
	for (it=0; it<iterations; it++){
		element=(num_lines*num_samples*blockIdx.x)+(blockDim.x*it);
		if((it*blockDim.x)+tid<num_lines*num_samples){
			sdata[tid]=(d_image[element+tid]-smean[0])*(d_image[element+tid]-smean[0]);
		}
		else{
			sdata[tid]=0;
		}
		__syncthreads();
		
		for(s=blockDim.x/2; s>0; s=s/2){
			if (tid < s){
				sdata[tid]+=sdata[tid+s];
			}
			__syncthreads();
		}
		
		if(tid==0){
			svar[0]+=sdata[0];
		}
		__syncthreads();
				
	}
	if(tid==0){
		svar[0]/=((num_lines*num_samples)-1);
		//d_pixel[blockIdx.x]=svar[0];
		
	}
	__syncthreads();

	for (it=0; it<iterations; it++){
		element=(num_lines*num_samples*blockIdx.x)+(blockDim.x*it);
		if((it*blockDim.x)+tid<num_lines*num_samples){
			d_image[element+tid]/=svar[0];
		}
	}

}


////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
//											Kernels NFINDR							  //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////



/*Este kernel calcula el volumen conseguido al agregar cada pixel al conjunto de endmembers.
Realiza la multiplicaci�n de d_aux * d_HIM2x2, ademas calcula el valor absoluto de cada volumen.*/

/*This kernel compute the getting volume obtained by adding each pixel to the set of 
endmembers. It makes the multiplication d_VVolume = d_aux * d_HIM2x2, also it gets 
the absolute value of each volume. */
__global__ void VolumeCalculation(double *d_aux, double *d_HIM2x2, double *d_Vvolume, 
											 double tmp2,int lines_samples, int n_end){
	int idx =  blockDim.x * blockIdx.x+threadIdx.x;
	__shared__ double s_aux[N];
	double a;
	if (idx<lines_samples){
		if(threadIdx.x<n_end){
			s_aux[threadIdx.x]=d_aux[threadIdx.x];
		}
		syncthreads();
		a=0;
		for(int i=0; i<n_end; i++){
			a+=s_aux[i]*d_HIM2x2[i*lines_samples+idx];
		}
		a=a*tmp2;
		d_Vvolume[idx]=ABS(a);
	}
}

/*Este kernel obtiene los I volumenes mayores calculados en el kernel anterior siendo I el n�mero
de bloques con que se estructura el lanzamiento del kernel. Adem�s obtiene los �ndices de los pixel
que otienen dichos volumenes.*/

/*This kernel gets the I higher volumes obtained by VolumeCalculation kernel, 
where I is the number of blocks that we configure the kernel launch. Also gets the index 
of the pixels that get this volumes.*/


__global__ void Reduction_vol(double *d_Vvolume, double *d_volumenes, int *d_indices, int lines_samples){

	__shared__ double s_v[512];
	__shared__ int s_i[512];

	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

	if((i+blockDim.x)>=lines_samples){
			s_v[tid]=d_Vvolume[i];
			s_i[tid]=i;
	}
	else{
		if(d_Vvolume[i]>d_Vvolume[i + blockDim.x]){
			s_v[tid]=d_Vvolume[i];
			s_i[tid]=i;
		}
		else{
			s_v[tid]=d_Vvolume[i + blockDim.x];
			s_i[tid]=i+ blockDim.x;
		}
	}
	__syncthreads();

	for (unsigned int s = blockDim.x / 2; s > 0; s>>=1){
		if (tid < s){
			if(s_v[tid]<=s_v[tid+s]){
				s_v[tid]=s_v[tid+s];
				s_i[tid]=s_i[tid+s];
			}
		}
		__syncthreads();
	}
	d_volumenes[blockIdx.x]=s_v[0];
	d_indices[blockIdx.x]=s_i[0];
	__syncthreads();
}

////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////
//											Kernels Unmixing						  //
////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////



/*Este kernel realiza la fase final del unmixing, es decir multiplicar cada pixel por la
matriz de c�mputo obtenida a partir de la matriz de endmembers, y as� obtener las abundancias.*/

/*This kernel multiplicates the compute_matrix by each pixel of the hyperspectral image, 
thus obtain a set of abundance vectors , each contain the fractional abundances of the 
endmembers in each pixel.*/
__global__ void Unmixing(float *d_imagen, float *d_imagen_unmixing,float *d_matriz_computo, int num_lines, int num_samples, int num_bands, int N_END)

{
	int pixel =  blockDim.x * blockIdx.x+threadIdx.x;
	
	__shared__ float matriz_c[TAMANIO_MATRIZ_C];
	float l_pixel[TAMANIO_MATRIZ_C];
	float a;
	if(pixel<num_lines*num_samples){
		for(int t=0; t<num_bands; t++){
			l_pixel[t]=d_imagen[pixel+(num_lines*num_samples*t)];
		}
		for(int it=0; it<N_END; it++){
			if(threadIdx.x==0){
				for(int i=0; i<num_bands; i++){
					matriz_c[i]=d_matriz_computo[it*num_bands+i];
				}
			}
			syncthreads();
			a=0;
			for(int k=0; k<num_bands; k++){	
				a+=matriz_c[k]*l_pixel[k];
			}
			d_imagen_unmixing[pixel+(num_lines*num_samples*it)]=a;
		}
	}

}
